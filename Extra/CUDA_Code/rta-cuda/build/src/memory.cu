/*
 
 memory.cpp
 
 Copyright (c) Michael Strickland
 
 GNU General Public License (GPLv3)
 See detailed text in license directory
 
 */

#include <stdlib.h>
#include <stdio.h>
#include <iostream>

using namespace std;

#include "rta.h"

double* allocate1DArray() {
    double *tmp;
    tmp = new double[num];
    return tmp;
}

// allocate flattened triangle array
double* allocateFTArray() {
    double *tmp;
    tmp = new double[num*(num+1)];
    return tmp;
}

double*** allocate3DArray(int n1, int n2, int n3) {
    double ***tmp;
    tmp = new double**[n1];
    for (int sx=0;sx<n1;sx++) tmp[sx] = new double*[n2];
    for (int sx=0;sx<n1;sx++) for (int sy=0;sy<n2;sy++) tmp[sx][sy] = new double[n3];
    return tmp;
}

void free1DArray(double *array) {
    delete[] array;
}

void free3DArray(double*** array, int n1, int n2, int n3) {
    for (int sx=0;sx<n1;sx++) for (int sy=0;sy<n2;sy++) free(array[sx][sy]);
    for (int sx=0;sx<n1;sx++) free(array[sx]);
    free(array);
    return;
}

void allocateMemory()
{
    cout << "==> Allocating memory\n";
    
    // allocate host memory
    t4 = allocate1DArray();
    T4 = allocate1DArray();
    t = allocate1DArray();
    hnm = allocateFTArray();
    hnm0 = allocate1DArray();
    f = allocate1DArray();
    
    // allocate device memory (GPU)
    hipMalloc((void**)&dev_t4, sizeof(double)*num);
    hipMalloc((void**)&dev_T4, sizeof(double)*num);
    hipMalloc((void**)&dev_time, sizeof(double)*num);
    hipMalloc((void**)&dev_d, sizeof(double)*num*(num+1)/2);
    hipMalloc((void**)&dev_h, sizeof(double)*num*(num+1)/2);
    hipMalloc((void**)&dev_hnm, sizeof(double)*num*(num+1)/2);
    hipMalloc((void**)&dev_hnm0, sizeof(double)*num);
    hipMalloc((void**)&dev_m, sizeof(double)*num*(num+1)/2);
    hipMalloc((void**)&dev_f, sizeof(double)*num);
}

void freeMemory()
{
    // free host memory
    free1DArray(t4);
    free1DArray(T4);
    free1DArray(t);
    free1DArray(hnm);
    free1DArray(hnm0);
    free1DArray(f);
    
    // free device memory
    hipFree( dev_t4 );
    hipFree( dev_T4 );
    hipFree( dev_time );
    hipFree( dev_d );
    hipFree( dev_h );
    hipFree( dev_hnm );
    hipFree( dev_hnm0 );
    hipFree( dev_m );
    hipFree( dev_f );
}

void swapPointers(double **pt1, double **pt2) {
    double *tmp = *pt1;
    *pt1 = *pt2;
    *pt2 = tmp;
}
