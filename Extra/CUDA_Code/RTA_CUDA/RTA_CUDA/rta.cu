#include "hip/hip_runtime.h"
/*
 
 rta.cu
 
 Copyright (c) Michael Strickland
 
 GNU General Public License (GPLv3)
 See detailed text in license directory
 
 */

#include <iostream>
#include <iomanip>
#include <fstream>
#include <cstdlib>
#include <cstdio>
#include <cmath>
#include <stdio.h>

#include <gsl/gsl_sf_hyperg.h>
#include <gsl/gsl_sf_gamma.h>

#include <hip/hip_runtime.h>
#include <>

#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

using namespace std;

#include "rta.h"
#include "outputroutines.h"
#include "paramreader.h"
#include "memory.h"

// defines
#define TIDX(i,j) (j + i*(i + 1)/2)
#define BLOCKSIZE1  256
#define BLOCKSIZE2  128

// constants that are shared with the GPU
__constant__ int NUM;
__constant__ double DTAU,A_0,T_0,EB;
__constant__ int N,M;
__constant__ double PZ,PT;
__constant__ double M_PI;
__constant__ double hbarc;

// these global vars are initialized from parameters file
// defaults set here are overridden by that file
int    num = 100, maxiters = 10, update = 10, snapupdate = 20;

double fpieb = 1; // 4 Pi eta / S
double t0 = 0.25; // initial time in fm/c
double tf = 20; // final time in fm/c
double T0 = 0.6; // initial temperature in GeV
double a0 = 1; // initial anisotropy a0 = 1/sqrt(1+xi0)


// time step
double dt;

// this holds the current values of T^4
double *t4;

// this holds the updated values of T^4
double *T4;

// this holds the integration abscissae (timeGrid)
double *t;

// parameters for moment computation
int computeMoments=0, maxN=4, maxM=4;

// this will hold the final solution for the distribution function f for a fix w and pt
double *f;
int computeDist=0,numPZ=40,numPT=40, fStep=1;;
double maxPT=2, maxPZ=2;

// these hold the values of hnm and the initial value array for the general moment equation
double *hnm,*hnm0;

// these are pointers for the device memory
double *dev_t4, *dev_T4, *dev_time, *dev_d, *dev_h, *dev_hnm, *dev_hnm0, *dev_m, *dev_f;

/*----------------------------------------------------------------------------------------------------*/
// Special functions
/*----------------------------------------------------------------------------------------------------*/

__device__ double H(double y) {
    if (y==1) return 2;
    if (fabs(y)<1) return y*(fabs(y) + asin(sqrt(1-y*y))/sqrt(1-y*y));
    if (fabs(y)>1) return y*(fabs(y) + asinh(sqrt(y*y-1))/sqrt(y*y-1));
    return 0;
}

double hostH(double y) {
    if (y==1) return 2;
    if (fabs(y)<1) return y*(fabs(y) + asin(sqrt(1-y*y))/sqrt(1-y*y));
    if (fabs(y)>1) return y*(fabs(y) + asinh(sqrt(y*y-1))/sqrt(y*y-1));
    return 0;
}

double my2F1(double a, double b, double c, double z)
{
	if (fabs(z)<=1) return gsl_sf_hyperg_2F1(a,b,c,z);
	if (z<-1) return pow(1-z,-a)*gsl_sf_hyperg_2F1(a,c-b,c,z/(z-1));
	else { cout << "mu2F1 err" << endl; exit(-1); }
}

double H(int n, int m, double y) {
    if (n==1) return 2*pow(y,2*m+1)/(2*m+1);
    if (y==0) return 0;
    if (y==1) return 2./(2*m+1);
    return 2*pow(y,2*m+1)*my2F1(0.5+m, 0.5*(1-n), 1.5+m, 1-y*y)/(2*m+1);
}

/*----------------------------------------------------------------------------------------------------*/
// Damping function
/*----------------------------------------------------------------------------------------------------*/

__device__ double D(int i2, int i1, double *lt4, double *lt) {
    if (i1==i2) return 1;
    double res = 0, w = 1;
    for (int j = i1; j <= i2; j++) {
        if (j==i1 || j==i2) w = 0.5;
        else w = 1.0;
        res += w*pow(lt4[j],0.25)*lt[j];
    }

    res *= DTAU/hbarc/EB/5.;
    return exp(-res);
}

/*----------------------------------------------------------------------------------------------------*/
// Device routines for T^4 iterative computation
/*----------------------------------------------------------------------------------------------------*/

// right hand side for t4 update
__device__ double rhs(int i, double *lt4, double *lt, double *ld, double *lh) {
    double res = 0;
    double w = 1;
    // second term
    if (i>0) {
        for (int ip = 0; ip <= i; ip++) {
            if (ip==0 || ip==i) w = 0.5;
            else w = 1.0;
            res += w*ld[TIDX(i,ip)]*lh[TIDX(i,ip)]*pow(lt4[ip],1.25)*lt[ip];
        }
        res *= DTAU/hbarc/EB/10.;
    }
    // first term
    res += ld[TIDX(i,0)]*pow(T_0,4.)*H(A_0*lt[0]/lt[i])/H(A_0);
    // return result
    return res;
}

// makes one iteration
__global__ void makeIteration(double *lt4, double *lT4, double *lt, double *ld, double *lh) {
    //printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid<NUM) {
        lT4[tid] = rhs(tid,lt4,lt,ld,lh);
        tid += blockDim.x * gridDim.x;
    }
}

// load damping function
__global__ void loadDampingFunction(double *lt4, double *lt, double *ld) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < NUM*(NUM+1)/2) {
        int row = floor(-0.5 + sqrt(0.25 + 2 * tid));
        int triangularNumber = row * (row + 1) / 2;
        int column = tid - triangularNumber;
        ld[tid] = D(row,column,lt4,lt);
        tid += blockDim.x * gridDim.x;
    }
}

// load H function
__global__ void loadHFunction(double *lt, double *lh) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < NUM*(NUM+1)/2) {
        int row = floor(-0.5 + sqrt(0.25 + 2 * tid));
        int triangularNumber = row * (row + 1) / 2;
        int column = tid - triangularNumber;
        lh[tid] = H(lt[column]/lt[row]);
        tid += blockDim.x * gridDim.x;
    }
}

/*----------------------------------------------------------------------------------------------------*/
// Device routines for general moment computation
/*----------------------------------------------------------------------------------------------------*/

// right hand side for mnm update
__device__ double rhsMNM(int i, double *lt4, double *lt, double *ld, double *lhnm, double *lhnm0, double lH0) {
    double res = 0;
    double w = 1;
    int r = N + 2*M + 2;
    // second term
    if (i>0) {
        for (int ip = 0; ip <= i; ip++) {
            if (ip==0 || ip==i) w = 0.5;
            else w = 1.0;
            res += w*ld[TIDX(i,ip)]*lhnm[TIDX(i,ip)]*pow(lt4[ip],0.25*(1+r))*lt[ip];
        }
        res *= DTAU/hbarc/EB/5.;
    }
    // first term
    res += pow(2.,0.25*r)*ld[TIDX(i,0)]*pow(T_0,r)*lhnm0[i]/pow(lH0,0.25*r);
    // return result
    return tgamma((double)r)*res/2/2/M_PI/M_PI;
}

// makes one iteration; this is a "kernel"
__global__ void computeMNM(double *lm, double *lt4, double *lt, double *ld, double *lhnm, double *lh, double *lhnm0, double H0) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid<NUM) {
        lm[tid] = rhsMNM(tid,lt4,lt,ld,lhnm,lhnm0,H0);
        tid += blockDim.x * gridDim.x;
    }
}

/*----------------------------------------------------------------------------------------------------*/
// Host routines for calculating f
/*----------------------------------------------------------------------------------------------------*/

// right hand side for f update
__device__ double rhsF(int i, double *lt4, double *lt, double *ld) {
    double res = 0, feq=0, T=1;
    double w = 1;
    // second term
    if (i>0) {
        for (int ip = 0; ip <= i; ip++) {
            if (ip==0 || ip==i) w = 0.5;
            else w = 1.0;
            T = pow(lt4[ip],0.25);
            feq = exp(-sqrt(PZ*PZ+PT*PT));
            res += w*ld[TIDX(i,ip)]*feq*T*lt[ip];
        }
        res *= DTAU/hbarc/EB/5.;
    }
    // first term
    T = pow(lt4[i],0.25);
    double l0 = pow(2./H(A_0),0.25)*T_0;
    double f0 = exp(-sqrt(pow(PZ*lt[i]/(A_0*lt[0]),2) + PT*PT)/(l0/T));
    res += ld[TIDX(i,0)]*f0;
    // return result
    return res;
}

// makes one iteration; this is a "kernel"
__global__ void computeF(double *lf, double *lt4, double *lt, double *ld) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid<NUM) {
        lf[tid] = rhsF(tid,lt4,lt,ld);
        tid += blockDim.x * gridDim.x;
    }
}

/*----------------------------------------------------------------------------------------------------*/
// Host routines for T^4 iterations
/*----------------------------------------------------------------------------------------------------*/

void makeIterations(double *lt4, double *lT4, double *lt, double *ld, double *lh) {

    outputMeasurements(0);
    outputTemperatureSnapshot(t4,0,"T");

    // load H function
    loadHFunction<<<num*(num+1)/2/BLOCKSIZE1,BLOCKSIZE1>>>(lt,lh);
    checkCudaErrors(hipDeviceSynchronize());

    for (int i=1; i<=maxiters;i++) {

        // load D function
        loadDampingFunction<<<num*(num+1)/2/BLOCKSIZE1,BLOCKSIZE1>>>(lt4,lt,ld);
        hipDeviceSynchronize();
        
        // make an iteration
        makeIteration<<<num/BLOCKSIZE2,BLOCKSIZE2>>>(lt4,lT4,lt,ld,lh);
        hipDeviceSynchronize();
        
        // swap pointers to make old <-> new
        swapPointers(&lt4,&lT4);
        
        // output some stuff if appropriate
        if (i%update==0) {
            hipMemcpy(t4, lt4, num*sizeof(double), hipMemcpyDeviceToHost);
            outputMeasurements(i);
        }
        if (i%snapupdate==0) {
            hipMemcpy(t4, lt4, num*sizeof(double), hipMemcpyDeviceToHost);
            outputTemperatureSnapshot(t4,i,"T");
        }
    }
    // load the device d function based on final result and copy t4 back to host for subsequent use
    loadDampingFunction<<<num*(num+1)/2/BLOCKSIZE1,BLOCKSIZE1>>>(lt4,lt,ld);
    hipDeviceSynchronize();
    checkCudaErrors(hipMemcpy(t4, lt4, num*sizeof(double), hipMemcpyDeviceToHost));
}

// loads integration abscissae
void loadTimeGrid() {
    cout << "==> Loading time grid" << endl;
    double ltf = log(tf);
    double lt0 = log(t0);
    dt = (ltf-lt0)/(num-1);
    for (int i = 0; i < num; i++) t[i] = exp(lt0 + i*dt);
}

// initializes t4 array
void initializeT4() {
    cout << "==> Initializing T^4 array" << endl;
    t4[0] = T0*T0*T0*T0;
    for (int i=1; i < num; i++) {
        t4[i] = T0*T0*T0*T0*pow(t0/t[i],4./3.);
    }
}

/*----------------------------------------------------------------------------------------------------*/
// Host routines for general moment computation
/*----------------------------------------------------------------------------------------------------*/

// initializes hnm array
void setupHNM(int n, int m) {
    for (int idx=0; idx < num*(num+1)/2; idx++) {
        int row = floor(-0.5 + sqrt(0.25 + 2 * idx));
        int triangularNumber = row * (row + 1) / 2;
        int column = idx - triangularNumber;
        hnm[idx] = H(n,m,t[column]/t[row]);
    }
    for (int idx=0; idx < num; idx++)
        hnm0[idx] = H(n,m,t[0]*a0/t[idx]);
}

// computes a general moment based on the current iterations results for t4
double* computeMoment(int n, int m) {
    cout << "==> Computing M(" << n << "," << m << ")" << endl;

    hipMemcpyToSymbol(HIP_SYMBOL(&N), &n, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(&M), &m, sizeof(int));
    
    setupHNM(n,m);
    hipMemcpy(dev_hnm, hnm, sizeof(double)*num*(num+1)/2, hipMemcpyHostToDevice); // transfer to device
    hipMemcpy(dev_hnm0, hnm0, sizeof(double)*num, hipMemcpyHostToDevice); // transfer to device
    
    double *lm;
    lm = allocate1DArray();
    computeMNM<<<num/BLOCKSIZE2,BLOCKSIZE2>>>(dev_m, dev_t4, dev_time, dev_d, dev_hnm, dev_h, dev_hnm0, hostH(a0));
    hipMemcpy(lm, dev_m, num*sizeof(double), hipMemcpyDeviceToHost);
    
    return lm;
}

// computes a general moment based on an equilbrium form with t4
inline double computeEQMoment(int n, int m, int i) {
    int r = n+2*m+2;
    return gsl_sf_gamma(r)*pow(t4[i],0.25*r)*2/(2*m+1)/2/2/M_PI/M_PI;
}

/*----------------------------------------------------------------------------------------------------*/
// Host routines for f computation
/*----------------------------------------------------------------------------------------------------*/

// computes f based on the current iterations results for t4
double* computeDistributionFunction(double pz, double pt) {
    //cout << "==> Computing f(" << pz << "," << pt << ")" << endl;
    
    hipMemcpyToSymbol(HIP_SYMBOL(&PZ), &pz, sizeof(double));
    hipMemcpyToSymbol(HIP_SYMBOL(&PT), &pt, sizeof(double));

    double *lf;
    lf = allocate1DArray();
    computeF<<<num/BLOCKSIZE2,BLOCKSIZE2>>>(dev_f, dev_t4, dev_time, dev_d);
    hipMemcpy(lf, dev_f, num*sizeof(double), hipMemcpyDeviceToHost);
    
    return lf;
}

/*----------------------------------------------------------------------------------------------------*/
// Main routine
/*----------------------------------------------------------------------------------------------------*/

int main(int argc, char** argv) {
    const double m_pi = 4.0 * atan(1.0);

    char fname[20]; // for later use
    
    print_line();
    
    // read parameters from file and command line
    readParametersFromFile("params.txt",1);
    if (argc>1) {
        print_line();
        cout << "Parameters from commandline" << endl;
        print_line();
        readParametersFromCommandLine(argc,argv,1);
    }
    // perform any processing of parameters necessary
    processParameters();
    
    print_line();
    print_line();
    print_line();
    
    //setup
    allocateMemory();
    loadTimeGrid();
    initializeT4();
        
    print_line();
    // copy grid and initial conditions to device
    checkCudaErrors(hipMemcpy(dev_t4, t4, sizeof(double)*num, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dev_T4, T4, sizeof(double)*num, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dev_time, t, sizeof(double)*num, hipMemcpyHostToDevice));

    print_line();
    // copy parameters to device memory
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(NUM), &num, sizeof(int)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(A_0), &a0, sizeof(double)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(T_0), &T0, sizeof(double)));
    double eb = fpieb/m_pi/4.;
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(EB), &eb, sizeof(double)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(DTAU), &dt, sizeof(double)));

    print_line();
    // Copy to grid global constants
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(M_PI), &m_pi, sizeof(double)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(hbarc), &HBARC, sizeof(double)));
    
    // print some stuff
    print_line();
    cout.width(dwidth); cout << "iteration";
    cout.width(dwidth); cout << "T[0]";
    cout.width(dwidth); cout << "T[num/2]";
    cout.width(dwidth); cout << "T[num-1]";
    cout << endl;
    print_line();
    
    /*----------------------------------------------------------------------------------------------------*/
    // Iterations
    /*----------------------------------------------------------------------------------------------------*/

    cout << "4 pi eta / S: " << fpieb << endl;
    makeIterations(dev_t4, dev_T4, dev_time, dev_d, dev_h);
    
    /*----------------------------------------------------------------------------------------------------*/
    // Compute some things with the solution
    /*----------------------------------------------------------------------------------------------------*/

    print_line();
    double *ed, *pl,*pt,*plopt;
    ed = allocate1DArray();
    pl = computeMoment(0,1);
    pt = allocate1DArray();
    plopt = allocate1DArray();
    for (int i=0; i<num; i++) {
        ed[i] = 3*t4[i]/m_pi/m_pi;
        pt[i] = 0.5*(ed[i] - pl[i]);
        plopt[i] = pl[i]/pt[i];
        cout << ed[i] << "\t" << pl[i] << "\t" << pt[i] << endl;
    }

    outputArray(ed,"ed");
    outputArray(pl,"pl");
    outputArray(pt,"pt");
    outputArray(plopt,"pratio");


    // compute distribution function
    if (computeDist==1) {
        print_line();
        cout << "==> Computing f ";
        double ***f3DArray;
        f3DArray = allocate3DArray(num/fStep,numPZ,numPT);
        double dpz = maxPZ/(numPZ-1);
        double dpt = maxPT/(numPT-1);
        for (int i=0; i<numPZ; i++) {
            for (int j=0; j<numPT; j++) {
                double *f;
                f = computeDistributionFunction(i*dpz,j*dpt);
                for (int k=0; k<num/fStep; k++) f3DArray[k][i][j] = f[k*fStep]; // load into f array for later binary output
                free1DArray(f);
            }
            cout << "." << std::flush;
        }
        cout << endl;
        outputDistribution(f3DArray); // output f in binary format
        free3DArray(f3DArray,num/fStep,numPZ,numPT);
    }
    
    if (computeMoments==1) {
        // loop over moments
        print_line();
        for (int n=0; n<=maxN; n++) {
            for (int m=0; m<=maxM; m++) {
                double *mom;
                mom = computeMoment(n,m);
                sprintf(fname,"moms/m-%d-%d",n,m);
                outputArray(mom,fname);
                for (int i=0; i<num; i++) mom[i] /= computeEQMoment(n,m,i);
                sprintf(fname,"moms/m-%d-%d-scaled",n,m);
                outputScaledArray(mom,t4,5*eb,fname);
                free1DArray(mom);
            }
        }
    }
    
    free1DArray(ed);
    free1DArray(pl);
    free1DArray(pt);
    free1DArray(plopt);

    /*----------------------------------------------------------------------------------------------------*/

    // print some more stuff
    print_line();
    cout << "Done.\n";
    print_line();
    
    // free memory
    freeMemory();
    
    return 0;
}
